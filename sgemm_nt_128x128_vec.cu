
#include <hip/hip_runtime.h>
extern "C" __global__ __launch_bounds__(256) void sgemm_nt_128x128_vec(
  const float *param_A,
  const float *param_B,
  float *param_C,
  float param_alpha,
  float param_beta,
  int param_lda8,
  int param_ldb8,
  int param_ldc,
  int param_m,
  int param_n,
  int param_k) {
  __shared__ float share[128 * 8 * 4 + 32];
  int tid = threadIdx.x;
  share[tid] = 1;
}
