#include <string>
#include <map>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>


std::map<std::string, hipFunction_t> functions;
std::vector<hipModule_t> modules;

using namespace std;

void load_kernels() {
  const size_t kernel_size = 1;
  const string kernel_name[kernel_size] = {
    "sgemm_tn_128x128_vec",
  };

  for (size_t i = 0; i < kernel_size; ++i) {
    const string& name = kernel_name[i];
    const string path = name + ".cubin";

    hipModule_t module;
    hipFunction_t function;
    hipError_t res;

    // load module
    res = hipModuleLoad(&module, path.c_str());
    if (res != hipSuccess) {
      std::cerr << "Failed to load module: " << name << std::endl;
      exit(1);
    }

    // load function
    res = hipModuleGetFunction(&function, module, name.c_str());
    if (res != hipSuccess) {
      std::cerr << "Failed to load function: " << name << std::endl;
      exit(1);
    }

    functions[name] = function;
    modules.push_back(module);
  }
}

void sgemm_tn(float *A, float *B, float *C, int size) {
  float alpha = 1.0;
  float beta = 0.0;
  int gridA = size / 128 + (size % 128 != 0);
  int gridB = size / 128 + (size % 128 != 0);
  int lda = size * 32;
  int ldb = size * 32;
  int ldc = size;
  void *args[11] = {&A, &B, &C, &alpha, &beta, &lda, &ldb, &ldc, &size, &size, &size};
  hipError_t res = hipModuleLaunchKernel(functions["sgemm_tn_128x128_vec"], 1, gridA, gridB, 256, 1, 1, 0, 0, args, NULL); 
  if (res != hipSuccess) {
    std::cerr << "Error launching kernel " << res << std::endl;
    exit(1);
  }
}

void sgemm_nn(float *A, float *B, float *C, int size) {
  float alpha = 1.0;
  float beta = 0.0;
  int gridA = size / 128 + (size % 128 != 0);
  int gridB = size / 128 + (size % 128 != 0);
  int lda = size;
  int ldb = size * 32;
  int ldc = size;
  void *args[11] = {&A, &B, &C, &alpha, &beta, &lda, &ldb, &ldc, &size, &size, &size};
  hipError_t res = hipModuleLaunchKernel(functions["sgemm_nn_128x128_vec"], 1, gridA, gridB, 256, 1, 1, 0, 0, args, NULL); 
  if (res != hipSuccess) {
    std::cerr << "Error launching kernel " << res << std::endl;
    exit(1);
  }
}

void sgemm_nt(float *A, float *B, float *C, int size) {
  float alpha = 1.0;
  float beta = 0.0;
  int gridA = size / 128 + (size % 128 != 0);
  int gridB = size / 128 + (size % 128 != 0);
  int lda = size;
  int ldb = size;
  int ldc = size;
  void *args[11] = {&A, &B, &C, &alpha, &beta, &lda, &ldb, &ldc, &size, &size, &size};
  hipError_t res = hipModuleLaunchKernel(functions["sgemm_nt_128x128_vec"], 1, gridA, gridB, 256, 1, 1, 0, 0, args, NULL); 
  if (res != hipSuccess) {
    std::cerr << "Error launching kernel " << res << std::endl;
    exit(1);
  }
}

int main(int argc, char *argv[]) {
  hipFree(0);
  float *h_A, *h_B, *h_C;
  float *d_A, *d_B, *d_C;
  int size = 4096;
  h_A = (float *)malloc(size * size * sizeof(float));
  h_B = (float *)malloc(size * size * sizeof(float));
  h_C = (float *)malloc(size * size * sizeof(float));
  for (size_t i = 0; i < size * size; ++i) {
    h_A[i] = 1;
    h_B[i] = 1;
  }
  hipMalloc((void **)&d_A, sizeof(float) * size * size);
  hipMalloc((void **)&d_B, sizeof(float) * size * size);
  hipMalloc((void **)&d_C, sizeof(float) * size * size);
  hipMemcpy(d_A, h_A, sizeof(float) * size * size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeof(float) * size * size, hipMemcpyHostToDevice);
  load_kernels();
  std::string kernel = std::string(argv[0]);
  if (kernel == "tn") {
    sgemm_tn(d_A, d_B, d_C, size);
  } else if (kernel == "nn") {
    sgemm_nn(d_A, d_B, d_C, size);
  } else if (kernel == "nt") {
    sgemm_nt(d_A, d_B, d_C, size);
  } else {
    std::cerr << "tt kernel not supported: " << std::endl;
    exit(1);
  }
  hipMemcpy(h_C, d_C, sizeof(float) * size * size, hipMemcpyDeviceToHost);
  for (size_t i = 0; i < size * size; ++i) {
    if (h_C[i] != size) {
      std::cerr << "Error: " << i << ":" << h_C[i] << std::endl;
      exit(1);
    }
  }
  free(h_A);
  free(h_B);
  free(h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  // run successfully
  std::cout << "finish" << std::endl;
  return 0;
}
